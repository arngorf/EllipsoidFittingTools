#include <iostream>
#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <time.h>
int timeval_subtract(
    struct timeval* result, struct timeval* t2,struct timeval* t1) {
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) -
    (t1->tv_usec + resolution * t1->tv_sec) ;
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}

__global__
void functionKernel(float* d_in, float *d_out) {
    const unsigned int lid = threadIdx.x; // local id inside a block
    const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
    float val = d_in[gid]/(d_in[gid]-2.3);
    d_out[gid] = val*val*val;
}

class CudaInterface {
public:

    CudaInterface();

    void serialVsParallelTest();
private:

    int n;
    int memSize;

    float* inputArray;
    float* serialResult;
    float* parallelResult;
};

CudaInterface::CudaInterface() {

    // Test parameters
    n = 753411;

    // Store of memory
    memSize = n * sizeof(float);

    // Allocate CPU memory
    inputArray = (float*) malloc(memSize);
    serialResult = (float*) malloc(memSize);
    parallelResult = (float*) malloc(memSize);

    // Initialize input array
    for (int i = 1; i <= n; ++i) {
        inputArray[i] = (float)i;
    }
}

void CudaInterface::serialVsParallelTest() {

    // Allocate GPU memory
    float* d_in;
    float* d_out;

    hipMalloc((void**)&d_in, memSize);
    hipMalloc((void**)&d_out, memSize);

    // copy host memory to device
    hipMemcpy(d_in, inputArray, memSize, hipMemcpyHostToDevice);
    // execute the kernel

    unsigned int num_threads = n;
    unsigned int mem_size = num_threads*sizeof(float);
    unsigned int block_size = 256;
    unsigned int num_blocks = ((num_threads + (block_size - 1)) / block_size);


    // Begin timing
    unsigned long int elapsed;

    struct timeval t_start_p, t_end_p, t_diff_p;
    gettimeofday(&t_start_p, NULL);

    functionKernel<<<num_blocks, block_size>>>(d_in, d_out);

    gettimeofday(&t_end_p, NULL);
    timeval_subtract(&t_diff_p, &t_end_p, &t_start_p);
    elapsed = t_diff_p.tv_sec*1e6+t_diff_p.tv_usec;
    printf("Parallel execution took %d microseconds (%.2fms)\n",elapsed,elapsed/1000.0);

    //functionKernel<<<1, n>>>(d_in, d_out);
    // copy result from device to host
    hipMemcpy(parallelResult, d_out, memSize, hipMemcpyDeviceToHost);
    // print result

    hipFree(d_in);
    hipFree(d_out);

    struct timeval t_start_s, t_end_s, t_diff_s;
    gettimeofday(&t_start_s, NULL);

    for (int i = 0; i < n; ++i) {
        float val = inputArray[i]/(inputArray[i]-2.3);
        serialResult[i] = val*val*val;
    }

    gettimeofday(&t_end_s, NULL);
    timeval_subtract(&t_diff_s, &t_end_s, &t_start_s);
    elapsed = t_diff_s.tv_sec*1e6+t_diff_s.tv_usec;
    printf("Parallel execution took %d microseconds (%.2fms)\n",elapsed,elapsed/1000.0);

    for (int i = 0; i < 100; ++i) {
        std::cout << inputArray[i] << ", " << serialResult[i] << ", " << parallelResult[i] << std::endl;
    }

    free(inputArray);
    free(serialResult);
    free(parallelResult);

}

void cudaTest() {
    CudaInterface cudaInterface;
    cudaInterface.serialVsParallelTest();
}

int main() {

    cudaTest();

    return 0;
}